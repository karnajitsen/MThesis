
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdint.h>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
#include <malloc.h>
#include <iomanip>
using namespace std;
#define REPEAT 1
#define STRIDE 1
#define CACHELINE 8
#define ALLIGNMENT 64
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi);


int gcf(int a, int b)
{
	if (a == 0) return b;
	return gcf(b % a, a);
}


int main(int argc, char* argv[])
{
   if(argc != 3)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   int stride = atoi(argv[2]);
   unsigned long long *d_time, h_time;
   double tottime;
   Dtype *xj, *xi;
   Dtype *h_A, **d_A;
   int *d_N;           
   std::ofstream fp;
   srand (time(NULL));
   
    fp.open("/home/hpc/ihpc/ihpc002h/gpu-exp/mThesis/exp5/data/result.txt", std::ofstream::app);
 
    h_A  = (Dtype*)memalign(ALLIGNMENT,(N+2)*sizeof(Dtype)); 
     
    
    hipMalloc(&d_A, (N+2)*sizeof(Dtype));
    hipMalloc(&d_time, sizeof(unsigned long long));
    hipMalloc(&xj, sizeof(Dtype));
    hipMalloc(&xi, sizeof(Dtype));
    hipMalloc(&d_N, sizeof(int));    
        
    //int step = gcf (STRIDE, N);
    
     for(unsigned int i=0; i < N ; i++)
    {
     // stride = rand()%20;
      
      h_A[i] = ((Dtype)(uintptr_t)d_A) +  ( (i + stride) % N)*sizeof(Dtype);
    //  h_A[i] = i+1;
    }
    
    
    h_A[N]=0.0;
    h_A[N+1]=0.0;         
    tottime = 0.0;
    
    for (int i=0; i < 10; i++)
    {
    
     dim3 Db = dim3(1);
     dim3 Dg = dim3(1,1,1);
    hipMemcpy(d_A, h_A, (N+2)*sizeof(Dtype), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
   
    VecAdd<<<Db,Dg>>>(d_A, d_N, d_time, xj, xi);    
          
    hipDeviceSynchronize();
    hipMemcpy(&h_time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
   // cout << h_time << endl;
    tottime += (double) h_time;
    }
  //  cout << std::fixed << std::setprecision(6) << tottime << std::endl;
    tottime = tottime / (10240.0);
    fp << N*8.0/1024.0 << " " << tottime << std::endl;
   // std::cout << std::setprecision(2);
  //  cout << std::fixed << std::setprecision(6) << tottime << std::endl;
    
    for(int i =0; i < N ; i++)
    {
     //printf("%f ",(h_A[i]));
    }
   
    hipFree(d_A);
    hipFree(d_time);
    free(h_A);
    fp.close();
}
