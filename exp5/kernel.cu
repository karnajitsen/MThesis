#include "hip/hip_runtime.h"
#include<stdio.h>
#include "repeat.h"
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
#define REPEAT 10
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *A;   
   unsigned int start_t, end_t;  
  
   for (int it=0; it < *N; it++)   
    j=*(Dtype **)j;
     
    *xi=*j;   
   *d_time = 0;
      
    for (int it=0; it < 4; it++) 
    {
       start_t = clock();
       repeat256(j=*(Dtype **)(unsigned long long)j;) 
       end_t = clock();
       ((Dtype *)A)[*N]=(Dtype) j;
       *d_time += (unsigned long long)(end_t - start_t);
       
    }
   
     
    // ((Dtype *)A)[*N]=(Dtype) j;
     //*d_time = (unsigned long long)(end_t - start_t); 
    // printf( "%d %llu\n", *N, *d_time/256/4);
   
}

