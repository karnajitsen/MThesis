#include "hip/hip_runtime.h"
#include<stdio.h>
#include "repeat.h"
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
#define REPEAT 10
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *A;   
   unsigned int start_t, end_t;
   
   //for (int it=0; it < *N; it++)   
     //j=*(Dtype **)j;
     
    //*xi=*j;   
   
     
    for (int it=0; it < *N; it++) 
    {
       start_t = clock(); 
       j=*(Dtype **)(unsigned long long)j;
       end_t = clock();
       *d_time = (unsigned long long)(end_t - start_t); 
        printf( "%d %llu\n", *N, *d_time);
       
    }
   
     ((Dtype *)A)[*N]=(Dtype) j;
     //((Dtype *)A)[*N]=(Dtype) temp;

   
}

