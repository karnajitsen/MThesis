#include "hip/hip_runtime.h"
#include<stdio.h>
#include "repeat.h"
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
#define REPEAT 10
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *A;   
   unsigned int start_t, end_t;
  // int a=0;
   for (int it=0; it < *N; it++)   
     j=*(Dtype **)j;
     
    *xi=*j;   
   
     start_t = clock(); 
    // for (int it=0; it < *N; it++) 
    //{
       repeat2048(j=*(Dtype **)(unsigned long long)j;)
       repeat2048(j=*(Dtype **)(unsigned long long)j;)
      // printf("hello");
       //a+= *N;
    //}
     //repeat2048(j=*(Dtype **)(unsigned long long)j;)
     //repeat2048(j=*(Dtype **)(unsigned long long)j;)
    // printf("%llu***\n",j);
     //}
     end_t = clock();
      ((Dtype *)A)[*N]=(Dtype) j;
    //printf("Hello"); 
   *d_time = (unsigned long long)(end_t - start_t) /(2*2048); 
   //printf( "%d %llu %u %u %d\n", *N, *d_time, start_t,  end_t,a);
   
}

