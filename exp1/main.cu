
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdint.h>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
#include <malloc.h>
using namespace std;
#define REPEAT 1
#define STRIDE 1
#define CACHELINE 8
#define ALLIGNMENT 64
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi);


int gcf(int a, int b)
{
	if (a == 0) return b;
	return gcf(b % a, a);
}


int main(int argc, char* argv[])
{
   if(argc != 3)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   int stride = atoi(argv[2]);
   unsigned long long *d_time, h_time;
   Dtype *xj, *xi;
   Dtype *h_A, **d_A;
   int *d_N;           
   std::ofstream fp;
   srand (time(NULL));
   
    fp.open("/home/hpc/ihpc/ihpc002h/gpu-exp/mThesis/exp1/data/result.txt", std::ofstream::app);
 
    h_A  = (Dtype*)memalign(ALLIGNMENT,(N+2)*sizeof(Dtype)); 
     
    
    hipMalloc(&d_A, (N+2)*sizeof(Dtype));
    hipMalloc(&d_time, sizeof(unsigned long long));
    hipMalloc(&xj, sizeof(Dtype));
    hipMalloc(&xi, sizeof(Dtype));
    hipMalloc(&d_N, sizeof(int));    
        
    //int step = gcf (STRIDE, N);
    
     for(unsigned int i=0; i < N ; i++)
    {
      //stride = rand()%20;
      
      h_A[i] = ((Dtype)(uintptr_t)d_A) +  ( (i + stride) % N)*sizeof(Dtype);
    }
    
    h_A[N]=0.0;
    h_A[N+1]=0.0;         
   
    hipMemcpy(d_A, h_A, (N+2)*sizeof(Dtype), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
   
    VecAdd<<<1,1>>>(d_A, d_N, d_time, xj, xi);    
    
    hipMemcpy(&h_time, d_time, sizeof(double), hipMemcpyDeviceToHost);
  
    hipDeviceSynchronize();
   
    fp << N*8.0/1024.0 << " " << h_time << std::endl;
    
    for(int i =0; i < N ; i++)
    {
     //printf("%f ",(h_A[i]));
    }
   
    hipFree(d_A);
    hipFree(d_time);
    free(h_A);
    fp.close();
}
