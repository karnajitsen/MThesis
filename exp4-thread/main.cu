
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdint.h>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>
#include <sstream>
using namespace std;
#define REPEAT 1
#define STRIDE 1
#define CACHELINE 8
#define ALLIGNMENT 64
typedef unsigned long long Dtype;


__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi);


int gcf(int a, int b)
{
	if (a == 0) return b;
	return gcf(b % a, a);
}


int main(int argc, char* argv[])
{
   if(argc != 4)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   int stride = atoi(argv[2]);
   int noofthreads = atoi(argv[3]);
   unsigned long long *d_time, h_time;
   Dtype *xj, *xi;
   Dtype *h_A, **d_A;
   int *d_N;           
   std::ofstream fp;
   std::ostringstream fpath;
   // int i=system("pwd");
    string dpath = "/home/hpc/ihpc/ihpc002h/gpu-exp/Master-thesis/exp4-thread/data/result.txt";
    
    //fpath << i + dpath;
    
    //cout << fpath << " ***** ";
    fp.open(dpath.c_str(), std::ofstream::app);
 
    h_A  = (Dtype*)memalign(ALLIGNMENT,(N+2)*sizeof(Dtype)); 
     
    
    hipMalloc(&d_A, (N+2)*sizeof(Dtype));
    hipMalloc(&d_time, sizeof(unsigned long long));
    hipMalloc(&xj, sizeof(Dtype));
    hipMalloc(&xi, sizeof(Dtype));
    hipMalloc(&d_N, sizeof(int));    
        
    //int step = gcf (STRIDE, N);
    
     for(unsigned int i=0; i < N ; i++)
    {
      h_A[i] = ((Dtype)(uintptr_t)d_A) +  ( (i + stride) % N)*sizeof(Dtype);
    }
    
    h_A[N]=0;
    h_A[N+1]=0;         
    
    hipMemcpy(d_A, h_A, (N+2)*sizeof(Dtype), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
    
    VecAdd<<<1,noofthreads>>>(d_A, d_N, d_time, xj, xi);    
    
    hipMemcpy(&h_time, d_time, sizeof(double), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    
    fp << noofthreads << " " << h_time << std::endl;
    
    for(int i =0; i < N ; i++)
    {
    //  printf("%llu ",*(h_A[i]));
    }
   
    hipFree(d_A);
    free(h_A);
    fp.close();
}
