#include "hip/hip_runtime.h"
#include<time.h>
#include<stdio.h>
#include<iostream>
#define BLOCK 16
#define Real unsigned long long
using namespace std;

__global__ void VecAdd(Real* A, int* N, unsigned long long* d_time, int *xj, int *xi)
{
   int j = 0;
   unsigned int start_t, end_t;
   int iteration=BLOCK;
     
   for (int it=0; it < *N; it++)   
     j=A[j];
   
   *xi=j; 
  //for(int i = 0; i< *N/BLOCK; i++)
   //{
    start_t = clock();
    //for (int repeat=0; repeat < 5; repeat++)
    //{
    // j=0;
    for (int it=0; it < iteration; it++)   
    j=A[j];
    //} 
   end_t = clock();
   iteration += BLOCK;
   *(d_time+i) = (unsigned long long)(end_t - start_t)/(iteration*5) ; 
   printf( "%llu %d\n", *(d_time+i),j);
   //}
   *xj=j;
}

