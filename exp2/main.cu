#include "hip/hip_runtime.h"
#include<iostream>
#include<algorithm>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
#include <malloc.h>
using namespace std;
#define REPEAT 1
#define Real unsigned int
#define STRIDE 1
#define CACHELINE 8
#define ALLIGNMENT 64
#define BLOCK 16
__global__ void VecAdd(Real** A, int* N, unsigned long long* d_time, int* xj, int* xi);


int main(int argc, char* argv[])
{
   if(argc != 2)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   //register long elapsed=0;
   Real **h_A, **d_A;
   int *d_N;// *B, *C, *d_A, *d_B, *d_C ;           
   std::ofstream fp;
   fp.open("/home/hpc/ihpc/ihpc002h/gpu-exp/Master-thesis/exp2/data/result.txt", std::ofstream::app);
    unsigned long long *d_time, *h_time;
    Real *xj;
    A  = (Real*)memalign(ALLIGNMENT,(N+2)*sizeof(Real)); 
    h_time = (unsigned long long *)memalign(ALLIGNMENT, N*sizeof(unsigned long long)/BLOCK);
    
    hipMalloc(&d_A, (N)*sizeof(Real));
    hipMalloc(&d_time, N*sizeof(unsigned long long)/BLOCK);
    hipMalloc(&xj, sizeof(int));
    hipMalloc(&d_N, sizeof(Real));
    
    
   for(int i=0; i < N ; i++)
    {
      A[i] = (i+STRIDE)%N;
     // B[i] = rand()%5;
      //C[i] = 0.0;
      
    }
    A[N]=0;
    
    hipMemcpy(d_A, h_A, (N+2)*sizeof(Real), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
    //hipMemcpy(d_C, C, N*sizeof(Real), hipMemcpyHostToDevice );
    
    //for(int i=0 ; i < REPEAT ; i++)
    //{
    //gettimeofday(&tv1, NULL);                  
    VecAdd<<<1,1>>>(d_A, d_N, d_time, xj);
    //gettimeofday(&tv2, NULL);
    //elapsed += ((tv2.tv_sec-tv1.tv_sec)*1000000.0 + (tv2.tv_usec-tv1.tv_usec));
    //}
    
    hipMemcpy(h_time, d_time,  N*sizeof(unsigned long long)/BLOCK, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
   //cout << N << std::endl;
    
    //double td = (N * 8.0)/1024;
    //cout << h_time << " " << CLOCKS_PER_SEC << std::endl;
    
    //fp << N*4.0/1024.0 << " " << h_time << std::endl;
    
    for(int i =0; i < N/BLOCK ; i++)
    {
      fp << (i+1)*BLOCK << " " << *(h_time+i) << std::endl;
    }
     //fs << std::endl;
   
    hipFree(d_A);
    free(A);
    //hipFree(d_B);
    //hipFree(d_C);
    fp.close();
}
