#include "hip/hip_runtime.h"
#include<stdio.h>
#include "repeat.h"
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
#define REPEAT 10
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *A;   
   Dtype **k;
   unsigned int start_t, end_t;  
   k = (Dtype **) malloc(sizeof(Dtype) * (1024 * (*N)));
   
   for (int it=0; it < *N; it++)   
    j=*(Dtype **)j;
     
    *xi=*j;   
   *d_time = 0;
      
    for (int it=0; it < (*N) * 10; it++) 
    {
       start_t = clock();
       repeat256(j=*(Dtype **)(unsigned long long)j;)
      // k[it] = j;
       end_t = clock();
       ((Dtype *)A)[*N]=(Dtype) j;
       *d_time += (unsigned long long)(end_t - start_t);
       
    }
   
     
    // ((Dtype *)A)[*N]=(Dtype) j;
     //*d_time = (unsigned long long)(end_t - start_t); 
    // printf( "%d %llu\n", *N, *d_time/256/4);
   
}

