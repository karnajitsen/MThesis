#include "hip/hip_runtime.h"
#include<stdio.h>
#include "repeat.h"
typedef unsigned long long Dtype;
//typedef double Dtype;
//typedef int Dtype;
#define REPEAT 10
__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *A;   
   unsigned int start_t, end_t;  
  
   //for (int it=0; it < *N; it++)   
    //j=*(Dtype **)j;
     
    //*xi=*j;   
   //for(int i = 0 ; i < 32; i++)
   //{
     start_t = clock(); 
    for (int it=0; it < *N * 2; it++) 
    {
       j=*(Dtype **)(unsigned long long)j;
    }
   
     end_t = clock();
     ((Dtype *)A)[*N]=(Dtype) j;
     *d_time = (unsigned long long)(end_t - start_t) ; 
    printf( "%d %f\n", *N, (double) (*d_time/(2.0 * (*N)))); 
       
   
   
   
}

