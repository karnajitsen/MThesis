#include "hip/hip_runtime.h"
#include<time.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "repeat.h"
typedef unsigned long long Dtype;


__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j;
     unsigned int start_t, end_t;
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
   //for (int it=0; it < *N; it++)   
     j=A[tid];
     
    *xi=*j;   
   
     start_t = clock(); 
    //for (int it=0; it < 1000; it++) 
     //{     
     //temp=j;
     j=A[tid];
     //}
     end_t = clock();
    // printf("%d %d \n", tid, sizeof(d_time));
   d_time[tid] = (unsigned long long)(end_t - start_t); 
   printf( "%llu %u %u \n", d_time[tid], start_t, end_t);
   *xj=*j;
}

