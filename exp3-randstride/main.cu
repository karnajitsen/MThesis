
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdint.h>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>
#include <sstream>
#include <time.h>
using namespace std;
#define REPEAT 1
#define STRIDE 1
#define CACHELINE 8
#define ALLIGNMENT 64
typedef unsigned long long Dtype;


__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi);


int gcf(int a, int b)
{
	if (a == 0) return b;
	return gcf(b % a, a);
}


int main(int argc, char* argv[])
{
   if(argc != 3)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   int stride = atoi(argv[2]);
   unsigned long long *d_time, h_time;
   Dtype *xj, *xi;
   Dtype *h_A, **d_A;
   int *d_N;           
   std::ofstream fp;
   std::ostringstream fpath;
   // int i=system("pwd");
    string dpath = "/home/hpc/ihpc/ihpc002h/gpu-exp/Master-thesis/exp3-randstride/data/result.txt";
    srand (time(NULL));
    //fpath << i + dpath;
    
    //cout << fpath << " ***** ";
    fp.open(dpath.c_str(), std::ofstream::app);
 
    h_A  = (Dtype*)memalign(ALLIGNMENT,(N+2)*sizeof(Dtype)); 
     
    
    hipMalloc(&d_A, (N+2)*sizeof(Dtype));
    hipMalloc(&d_time, sizeof(unsigned long long));
    hipMalloc(&xj, sizeof(Dtype));
    hipMalloc(&xi, sizeof(Dtype));
    hipMalloc(&d_N, sizeof(int));    
        
    //int step = gcf (STRIDE, N);
    
     for(unsigned int i=0; i < N ; i++)
    {
      
      stride = rand()%10;
      h_A[i] = ((Dtype)(uintptr_t)d_A) +  ( (i + stride) % N)*sizeof(Dtype);
      
    }
    
    h_A[N]=0;
    h_A[N+1]=0;         
    
    hipMemcpy(d_A, h_A, (N+2)*sizeof(Dtype), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
    
    VecAdd<<<1,1>>>(d_A, d_N, d_time, xj, xi);    
    
    hipMemcpy(&h_time, d_time, sizeof(double), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    
    fp << stride << " " << h_time << std::endl;
    
    // printf("\n%llu ",(h_A[0]));
    
    for(int i =0; i < N ; i++)
    {
     //printf("\n%llu ",(h_A[i]));
    }
   
    hipFree(d_A);
    free(h_A);
    fp.close();
}
