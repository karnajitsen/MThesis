#include<iostream>
#include <stdint.h>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
#include <malloc.h>
#include <string.h>
#include <sstream>
#include <time.h>
#include <hip/hip_runtime.h>
using namespace std;
#define REPEAT 1
#define STRIDE 1
#define CACHELINE 8
#define ALLIGNMENT 64
typedef unsigned long long Dtype;


__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi);


int gcf(int a, int b)
{
	if (a == 0) return b;
	return gcf(b % a, a);
}


int main(int argc, char* argv[])
{
   if(argc != 3)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   int stride = atoi(argv[2]);
   unsigned long long *d_time;
   unsigned long long *h_time;
   Dtype *xj, *xi;
   Dtype *h_A, **d_A;
   int *d_N;           
   std::ofstream fp;
   std::ostringstream fpath;
   // int i=system("pwd");
    string dpath = "/home/hpc/ihpc/ihpc002h/gpu-exp/mThesis/exp3-randstride/data/result.txt";
    srand (time(NULL));
    //fpath << i + dpath;
    
    //cout << fpath << " ***** ";
    fp.open(dpath.c_str(), std::ofstream::app);
 
    h_A  = (Dtype*)memalign(ALLIGNMENT,(N+2)*sizeof(Dtype)); 
    h_time = (unsigned long long *) malloc(N*sizeof(unsigned long long));
    
    hipMalloc(&d_A, (N+2)*sizeof(Dtype));
    hipGetErrorString(hipMalloc((void **)&d_time, N*sizeof(unsigned long long)));
    hipMalloc(&xj, sizeof(Dtype));
    hipMalloc(&xi, sizeof(Dtype));
    hipMalloc(&d_N, sizeof(int));    
        
    
    //int step = gcf (STRIDE, N);
    
     for(unsigned int i=0; i < N ; i++)
    {      
      stride = rand()%10;
      //stride = 1;
      h_A[i] = ((Dtype)(uintptr_t)d_A) +  ( (i + stride) % N)*sizeof(Dtype);
      
    }
    
    h_A[N]=0;
    h_A[N+1]=0;         
    
    hipMemcpy(d_A, h_A, (N+2)*sizeof(Dtype), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
    hipGetErrorString(hipMemcpy(d_time, h_time, N*sizeof(unsigned long long), hipMemcpyHostToDevice ));    
    VecAdd<<<((N-1)/32)+1,32>>>(d_A, d_N, d_time, xj, xi);
    hipDeviceSynchronize();
    hipGetErrorString(hipMemcpy(h_time, d_time, N*sizeof(unsigned long long), hipMemcpyDeviceToHost)) ;
    //cudaDeviceSynchronize();
    fp << N << " " << h_time[0] << std::endl;
    
    // printf("\n%llu ",(h_A[0]));
    
    //for(int i =0; i < N ; i++)
   // {
     printf("\n%llu ",h_time[0]);
   // }
   
    hipFree(d_A);
    free(h_A);
    hipFree(d_time);
    free(h_time);
    fp.close();
}
