#include "hip/hip_runtime.h"
#include<time.h>
#include<stdio.h>
typedef unsigned long long Dtype;


__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *A;
   Dtype *temp;
   unsigned int start_t, end_t;
   
   for (int it=0; it < *N; it++)   
     j=*(Dtype **)j;
     
    *xi=*j;   
   
     start_t = clock(); 
     //for (int it=0; it < *N; it++) 
    // {     
     //temp=j;
     j=*(Dtype **)j;   
    /* j=*(Dtype **)j; 
     j=*(Dtype **)j;
     j=*(Dtype **)j; 
     j=*(Dtype **)j; 
     j=*(Dtype **)j;   
     j=*(Dtype **)j; 
     j=*(Dtype **)j;
     j=*(Dtype **)j; 
     j=*(Dtype **)j; */
     //
    // *(Dtype**)temp=j;
    // printf("%llu***\n",j);
     }
     end_t = clock();
     
   *d_time = (unsigned long long)(end_t - start_t)/(*N); 
   printf( "%llu %u %u \n", *d_time, start_t, end_t);
   *xj=*j;
}

