
#include <hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
typedef unsigned long long Dtype;


__global__ void VecAdd(Dtype** A, int* N, unsigned long long* d_time, Dtype* xj, Dtype* xi)
{
   Dtype *j = *(A+*N-1);   
   unsigned int start_t, end_t;
   
   for (int it=0; it < *N; it++)   
     j=*(Dtype **)j;
     
    *xi=*j;   
   
     start_t = clock(); 
     for (int it=0; it < *N*5; it++) 
     {
     j=*(Dtype **)j;     
    // printf("%llu***\n",j);
     }
     end_t = clock();
     
   *d_time = (unsigned long long)(end_t - start_t)/(*N*5); 
   printf( "%llu %u %u ", *d_time, start_t, end_t);
   *xj=*j;
}

