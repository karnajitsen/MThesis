#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  L2 cache size: %d\n\n", prop.l2CacheSize);
    printf("  total Global Memory size: %d\n\n", prop.totalGlobalMem);
    printf("  Unified addressing: %d\n\n", prop.unifiedAddressing);
     printf("  memoryBusWidth: %d\n\n", prop.memoryBusWidth);
    printf(" memoryClockRate: %d\n\n", prop.memoryClockRate);
    printf(" pciBusID: %d\n\n", prop.pciBusID);
    printf(" pciDeviceID: %d\n\n", prop.pciDeviceID);
    printf(" pciDomainID: %d\n\n", prop.pciDomainID); 
    
    
  }
} 
